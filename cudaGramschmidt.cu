#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>

#include <hip/hip_runtime.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// Image structure
typedef struct 
{
	int width, height; // Image size
	int bytes_per_pixel; // 1 for grayscale image, 3 for rgb image
	unsigned long long total_bytes; // Total bytes in data, width * height * bytes_per_pixel
	unsigned char * data; // Image data - very large array of 8-bit values
	
	float mean, variance, stddev; // Image metrics, left unevaluated at the beginning
} image_t;

void alloc_image(image_t * image, int width, int height, int bytes_per_pixel) 
{
	// Allocate large chunk of memory for the image
	image->width = width;
	image->height = height;
	image->bytes_per_pixel = bytes_per_pixel;
	image->total_bytes = (unsigned long long) (width) * height * bytes_per_pixel;
	
	image->data = (unsigned char *) malloc(width * height * bytes_per_pixel);
	if (!image->data) {
		printf("Unable to allocate %llu bytes of memory!", image->total_bytes);
		exit(1); // Critical error for us
	}
	
	image->mean = 0.0f;
	image->variance = 0.0f;
	image->stddev = 0.0f;
	
	printf("%llu MiB allocated for the image\n",
	             (unsigned long long) (image->total_bytes + sizeof(image_t) + 1024 * 1024 - 1) / (1024 * 1024));
}

void dealloc_image(image_t * image) {
	// Free memory
	free(image->data);
	printf("%llu MiB deallocated\n",
	             (unsigned long long) (image->total_bytes + sizeof(image_t) + 1024 * 1024 - 1) / (1024 * 1024));
	
	image->data = NULL;
	image->width = 0;
	image->height = 0;
	image->bytes_per_pixel = 0;
	image->total_bytes = 0;
}

void clone_image(image_t * image, image_t * src) {
	// Create a full copy of an image src
	alloc_image(image, src->width, src->height, src->bytes_per_pixel);
	memcpy(image->data, src->data, src->total_bytes);
	
	image->mean = src->mean;
	image->variance = src->variance;
	image->stddev = src->stddev;
}

void read_image(image_t * image, const char * path) {
	FILE * fp = fopen(path, "rb");
	if (!fp) {
		printf("File %s not found or can't be opened! Exiting...", path); // Critical error for our program
		exit(1);
	}
	
	unsigned long off_set = 0, next_offset = 0, pos_count = 0, tag_value[25] = { 0 }, strip_offset_val = 0, strip_offset = 0;
	int i, j, k, tag_id[25] = { 0 }; //, tag_type[25] = { 0 }, tag_count[25] = { 0 };
	int tagCount = 0, img_Width = 0, img_Len = 0, IFD_count = 0;
	short buffer[4] = { 0 };
	fseek(fp, 4, SEEK_SET);
	for (i = 3; i >= 0; i--) {
		buffer[i] = getc(fp);
	}
	off_set = ((int)buffer[0] << 24) | ((int)buffer[1] << 16) | ((int)buffer[2] << 8) | ((int)buffer[3]);

	fseek(fp, off_set, SEEK_SET);
	k = 1;
	while (k) {
		IFD_count++;														  // No. of IFD's in this TIFF File
		buffer[1] = getc(fp);
		buffer[0] = getc(fp);
		tagCount = ((int)buffer[0] << 8) | ((int)buffer[1]);	 // No. of entries in an IFD
		for (i = 0; i < tagCount; i++) {									// Read all the entries of this IFD
			buffer[1] = getc(fp);
			buffer[0] = getc(fp);
			tag_id[i] = ((int)buffer[0] << 8) | ((int)buffer[1]);		 // Tag ID

			for (j = 1; j >= 0; j--) {
				buffer[j] = getc(fp);
			}

			for (j = 3; j >= 0; j--) {
				buffer[j] = getc(fp);
			}
			
				  // Gives no. of values for this Tag
			for (j = 3; j >= 0; j--) {
				buffer[j] = getc(fp);
			}
			tag_value[i] = ((int)buffer[0] << 24) | ((int)buffer[1] << 16) | ((int)buffer[2] << 8) | ((int)buffer[3]);
				  // Gets the value if the above count is 1, else offset of the starting value
			if (tag_id[i] == 256)												 // Tag ID 256 says about the image width
				img_Width = tag_value[i];
			if (tag_id[i] == 257)												 // Tag ID 257 says about the image length
				img_Len = tag_value[i];
			if (tag_id[i] == 273)												 // Tag ID 273 says about the offset which points to the offset of strips
				strip_offset_val = tag_value[i];
		}	

		alloc_image(image, img_Width, img_Len, 1);
	
		for (i = 0; i < img_Len; i++) {										  // Read the pixel values from image and store it in the matrix
			fseek(fp, (strip_offset_val + (i * 4)), SEEK_SET);				  // Move to the offset of the current strip's offset

			for (j = 3; j >= 0; j--) {
				buffer[j] = getc(fp);
			}
			strip_offset = ((int)buffer[0] << 24) | ((int)buffer[1] << 16) | ((int)buffer[2] << 8) | ((int)buffer[3]);
			fseek(fp, strip_offset, SEEK_SET);	 // Move to the offset of the current strip
			for (j = 0; j < img_Width; j++) {

				getc(fp);
				short tmp_c2 = getc(fp);
				image->data[i * img_Width + j] = tmp_c2;
			}
		}
		pos_count = ((off_set + 2) + (tagCount * 12));
		fseek(fp, pos_count, SEEK_SET);
		for (i = 3; i >= 0; i--) {
			buffer[i] = getc(fp);
		}
		next_offset = ((int)buffer[0] << 24) | ((int)buffer[1] << 16) | ((int)buffer[2] << 8) | ((int)buffer[3]); // Next IFD offset
		if (next_offset != 0)
		{
			fseek(fp, next_offset, SEEK_SET);
		}
		else {
			k = 0;
		}
	}
	
	fclose(fp);
	
	printf("Image %s loaded successfully\n", path);
}

void WriteHexString(FILE *fptr, char *s) {
   unsigned int i, c;
   char hex[3];

   for (i = 0; i<strlen(s); i += 2) {
      hex[0] = s[i]; 
      hex[1] = s[i + 1];
      hex[2] = '\0';
      sscanf(hex, "%X", &c);
      putc(c, fptr);
   }
}

void write_image(image_t * image, const char * path) 
{
	if ((image->bytes_per_pixel != 1) && (image->bytes_per_pixel != 3)) 
	{
		printf("Only 1 and 3 bytes per pixel images are supported in write_image procedure");
		exit(1);
		return;
	}
	
	FILE * fptr = fopen(path, "wb");
	if (!fptr) {
		printf("File %s can't be opened for writing! Exiting...", path); // Critical error for our program
		exit(1);
	}
	
   /* Write the header */
   WriteHexString(fptr, "4d4d002a");    /* Big endian & TIFF identifier */
   int nx = image->width;
   int ny = image->height;
   int offset = nx * ny * 3 + 8;
   putc((offset & 0xff000000) / 16777216, fptr);
   putc((offset & 0x00ff0000) / 65536, fptr);
   putc((offset & 0x0000ff00) / 256, fptr);
   putc((offset & 0x000000ff), fptr);

   /* Write the binary data */
	unsigned long long i;
	
	if (image->bytes_per_pixel == 3) {
		// Just save the data "as is"
		for (i = 0; i < image->total_bytes; i++)
			fputc(image->data[i], fptr);
	} else {
		// Save each pixel three times as r, g, b component
		for (i = 0; i < image->total_bytes; i++) {
			fputc(image->data[i], fptr);
			fputc(image->data[i], fptr);
			fputc(image->data[i], fptr);
		}
	}

   /* Write the footer */
   WriteHexString(fptr, "000e");  /* The number of directory entries (14) */

   /* Width tag, short int */
   WriteHexString(fptr, "0100000300000001");
   fputc((nx & 0xff00) / 256, fptr);    /* Image width */
   fputc((nx & 0x00ff), fptr);
   WriteHexString(fptr, "0000");

   /* Height tag, short int */
   WriteHexString(fptr, "0101000300000001");
   fputc((ny & 0xff00) / 256, fptr);    /* Image height */
   fputc((ny & 0x00ff), fptr);
   WriteHexString(fptr, "0000");

   /* Bits per sample tag, short int */
   WriteHexString(fptr, "0102000300000003");
   offset = nx * ny * 3 + 182;
   putc((offset & 0xff000000) / 16777216, fptr);
   putc((offset & 0x00ff0000) / 65536, fptr);
   putc((offset & 0x0000ff00) / 256, fptr);
   putc((offset & 0x000000ff), fptr);

   /* Compression flag, short int */
   WriteHexString(fptr, "010300030000000100010000");

   /* Photometric interpolation tag, short int */
   WriteHexString(fptr, "010600030000000100020000");

   /* Strip offset tag, long int */
   WriteHexString(fptr, "011100040000000100000008");

   /* Orientation flag, short int */
   WriteHexString(fptr, "011200030000000100010000");

   /* Sample per pixel tag, short int */
   WriteHexString(fptr, "011500030000000100030000");

   /* Rows per strip tag, short int */
   WriteHexString(fptr, "0116000300000001");
   fputc((ny & 0xff00) / 256, fptr);
   fputc((ny & 0x00ff), fptr);
   WriteHexString(fptr, "0000");

   /* Strip byte count flag, long int */
   WriteHexString(fptr, "0117000400000001");
   offset = nx * ny * 3;
   putc((offset & 0xff000000) / 16777216, fptr);
   putc((offset & 0x00ff0000) / 65536, fptr);
   putc((offset & 0x0000ff00) / 256, fptr);
   putc((offset & 0x000000ff), fptr);

   /* Minimum sample value flag, short int */
   WriteHexString(fptr, "0118000300000003");
   offset = nx * ny * 3 + 188;
   putc((offset & 0xff000000) / 16777216, fptr);
   putc((offset & 0x00ff0000) / 65536, fptr);
   putc((offset & 0x0000ff00) / 256, fptr);
   putc((offset & 0x000000ff), fptr);

   /* Maximum sample value tag, short int */
   WriteHexString(fptr, "0119000300000003");
   offset = nx * ny * 3 + 194;
   putc((offset & 0xff000000) / 16777216, fptr);
   putc((offset & 0x00ff0000) / 65536, fptr);
   putc((offset & 0x0000ff00) / 256, fptr);
   putc((offset & 0x000000ff), fptr);

   /* Planar configuration tag, short int */
   WriteHexString(fptr, "011c00030000000100010000");

   /* Sample format tag, short int */
   WriteHexString(fptr, "0153000300000003");
   offset = nx * ny * 3 + 200;
   putc((offset & 0xff000000) / 16777216, fptr);
   putc((offset & 0x00ff0000) / 65536, fptr);
   putc((offset & 0x0000ff00) / 256, fptr);
   putc((offset & 0x000000ff), fptr);

   /* End of the directory entry */
   WriteHexString(fptr, "00000000");

   /* Bits for each colour channel */
   WriteHexString(fptr, "000800080008");

   /* Minimum value for each component */
   WriteHexString(fptr, "000000000000");

   /* Maximum value per channel */
   WriteHexString(fptr, "00ff00ff00ff");

   /* Samples per pixel for each channel */
   WriteHexString(fptr, "000100010001");
	
	fclose(fptr);
	printf("File %s written successfully.\n", path);
}

__global__ void cuda_CovVar(const unsigned char *input1, const unsigned char *input2, float mean1, float mean2, const unsigned long long n, double *cov, double *var)
{
	__shared__ double sharedCovData[512];
	__shared__ double sharedVarData[512];

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = threadIdx.x;

	sharedCovData[tx] = 0;
	sharedVarData[tx] = 0;

	__syncthreads();

	for(unsigned long long i = id; i < n; i+= gridDim.x*blockDim.x) 
	{
		sharedCovData[tx] += ((double)input1[i]-mean1)*((double)input2[i]-mean2);
		sharedVarData[tx] += ((double)input2[i]-mean2)*((double)input2[i]-mean2);		
	}

	__syncthreads();

	// block-wide reduction in _shared_ mem
	for(int offset = blockDim.x / 2; offset > 0; offset >>= 1)
	{
		if(tx < offset)
		{
			sharedCovData[tx] += sharedCovData[tx + offset];
			sharedVarData[tx] += sharedVarData[tx + offset];
		}
		__syncthreads();
	}

	// finally, thread 0 writes the calculated result of this block
	if(threadIdx.x == 0)
	{
		// final result should be updated in an exclusive way by each block
		cov[blockIdx.x] = sharedCovData[0];
		var[blockIdx.x] = sharedVarData[0];

	}
}

__global__ void cuda_mean(const unsigned char *input, double *mean,  const unsigned long long n)
{
	__shared__ float sharedSumData[512];

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = threadIdx.x;
	sharedSumData[tx] = 0;

	for(unsigned long long i = id; i < n; i+= gridDim.x*blockDim.x) 
	{
		sharedSumData[tx] += (float)input[i];
	}

	__syncthreads();

	// block-wide reduction in _shared_ mem
	for(int offset = blockDim.x / 2; offset > 0; offset >>= 1)
	{
		if(tx < offset)
		{
			sharedSumData[tx] += sharedSumData[tx + offset];
		}
		__syncthreads();
	}

	// finally, thread 0 writes the calculated result of this block
	if(threadIdx.x == 0)
	{
		// final result should be updated in an exclusive way by each block
		mean[blockIdx.x] = sharedSumData[0];

	}
}

__global__ void cuda_var(const unsigned char *input, double mean,  const unsigned long long n, double *var)
{
	__shared__ float sharedSumData[512];

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = threadIdx.x;

	sharedSumData[tx] = 0;

	for(unsigned long long i = id; i < n; i+= gridDim.x*blockDim.x) 
	{
		sharedSumData[tx] += ((float)input[i]-mean)*((float)input[i]-mean);
	}
	//sharedSumData[tx] = (x-lmean)*(x-lmean);

	__syncthreads();

	// block-wide reduction in _shared_ mem
	for(int offset = blockDim.x / 2; offset > 0; offset >>= 1)
	{
		if(tx < offset)
		{
			sharedSumData[tx] += sharedSumData[tx + offset];
		}
		__syncthreads();
	}

	// finally, thread 0 writes the calculated result of this block
	if(threadIdx.x == 0)
	{
		// final result should be updated in an exclusive way by each block
		var[blockIdx.x] =  sharedSumData[0];

	}
}
// Evaluate image mean and standart deviation
void cuda_eval_stats(image_t * image, unsigned char *data ) 
{
	if (image->bytes_per_pixel != 1) {
		printf("Mean and standart deviation are only evaluated for grayscale images!\n");
		exit(1);
	}
	
	double mean = 0.0f, variance = 0.0f, stddev = 0.0f;
	unsigned long long i;
	
	// Evaluate mean
	//for (i = 0; i < image->total_bytes; i++)
	//	mean += image->data[i];
	double *d_mean;
	double h_mean[16];
	gpuErrchk(hipMalloc((void**)&d_mean, 16*sizeof(double)));	
	cuda_mean<<<16,256>>>(data, d_mean, image->total_bytes);
	hipMemcpy(h_mean, d_mean, 16*sizeof(double) , hipMemcpyDeviceToHost );
	hipFree(d_mean);

	for (i = 0; i < 16; i++)
		mean += h_mean[i];

	//printf("mean: %lf\n",mean);
	mean /= image->total_bytes;
	
	// Evaluate variance
	//for (i = 0; i < image->total_bytes; i++) {
	//	float tmp = (float) (image->data[i]) - mean;
	//	variance += tmp * tmp;
	//}
	double *d_var;
	double h_var[16];
	gpuErrchk(hipMalloc((void**)&d_var, 16*sizeof(double)));	
	cuda_var<<<16,256>>>(data, mean, image->total_bytes, d_var);
	hipMemcpy(h_var, d_var, 16*sizeof(double) , hipMemcpyDeviceToHost );
	hipFree(d_var);

	for (i = 0; i < 16; i++)
		variance += h_var[i];

	//printf("var: %lf\n",variance);


	//printf("variance: %f\n",mean);
	variance /= image->total_bytes;
	
	stddev = sqrtf(variance);
	
	// Set up those values
	image->mean = (float) mean;
	image->stddev = (float) stddev;
}

// Evaluate image mean and standart deviation
void eval_stats(image_t * image) 
{
	if (image->bytes_per_pixel != 1) {
		printf("Mean and standart deviation are only evaluated for grayscale images!\n");
		exit(1);
	}
	
	double mean = 0.0f, variance = 0.0f, stddev = 0.0f;
	unsigned long long i;
	
	// Evaluate mean
	for (i = 0; i < image->total_bytes; i++)
		mean += image->data[i];
	mean /= image->total_bytes;
	
	// Evaluate variance
	for (i = 0; i < image->total_bytes; i++) {
		float tmp = (float) (image->data[i]) - mean;
		variance += tmp * tmp;
	}
	variance /= image->total_bytes;
	
	stddev = sqrtf(variance);
	
	// Set up those values
	image->mean = mean;
	image->stddev = stddev;
}

float covariance(image_t * b, image_t * gs) 
{
	// Estimate mean
	//eval_stats(b);
	//eval_stats(gs);
	
	double covariance = 0.0f;
	double variance = 0.0f;
	
	unsigned long long i;
	//for (i = 0; i < b->total_bytes; i++) 
	//{
	//	covariance += (b->data[i] - b->mean) * (gs->data[i] - gs->mean);
	//	variance += (gs->data[i] - gs->mean) * (gs->data[i] - gs->mean);
	//}
	// printf("Covariance - Variance : %.2lf - %.2lf \n", covariance, variance);

	unsigned char *d_b, *d_gs;
	double *d_covariance, *d_variance;
	gpuErrchk(hipMalloc((void**)&d_variance, 16*sizeof(double)));
	gpuErrchk(hipMalloc((void**)&d_covariance, 16*sizeof(double)));
	gpuErrchk(hipMalloc((void**)&d_gs,  b->total_bytes*sizeof(unsigned char)));	
	gpuErrchk(hipMalloc((void**)&d_b,  b->total_bytes*sizeof(unsigned char)));
	gpuErrchk(hipMemcpy(d_gs, gs->data, b->total_bytes*sizeof(unsigned char), hipMemcpyHostToDevice) );
	gpuErrchk(hipMemcpy(d_b, b->data, b->total_bytes*sizeof(unsigned char), hipMemcpyHostToDevice) );

	cuda_eval_stats(b, d_b);
	cuda_eval_stats(gs, d_gs);


	cuda_CovVar<<< 16, 256>>>(d_b, d_gs, b->mean, gs->mean, b->total_bytes, d_covariance, d_variance);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	
	double covs[16], vars[16];
	hipMemcpy(covs, d_covariance, 16*sizeof(double) , hipMemcpyDeviceToHost );
	hipMemcpy(vars, d_variance, 16*sizeof(double) , hipMemcpyDeviceToHost );
	for (i = 0; i < 16; i++) 
	{
		covariance += covs[i];
		variance += vars[i];
	}

	//printf("Covariance - Variance : %.2lf - %.2lf \n", covariance, variance);

	// Both should be divided by N, but we're going to divide them anyway
	hipFree(d_gs);
	hipFree(d_b);
	hipFree(d_covariance);
	hipFree(d_variance);
	//free(covs);

	return covariance / variance;	
}




__global__ void cuda_GramSchmidt(float *results, unsigned long long n, const unsigned char * gs, float phi) 
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	for(int i = idx; i<n; i += gridDim.x*blockDim.x)
	{
		results[i] = results[i] - (phi * gs[i]);
	}
}

// Core routine - Gram-Schmidt transformation
void GramSchmidtTransformation(image_t * gs, image_t * bands, float ** phi) 
{
	// Gram-Schmidt imlementation for 4 vectors
	
	// First gs element is the same as band 0, i.e. artificial low res pan image
	clone_image(gs + 0, bands + 0);
	alloc_image(gs + 1, bands[0].width, bands[0].height, bands[0].bytes_per_pixel);
	alloc_image(gs + 2, bands[0].width, bands[0].height, bands[0].bytes_per_pixel);
	alloc_image(gs + 3, bands[0].width, bands[0].height, bands[0].bytes_per_pixel);
	
	float *res = (float *)malloc(bands[0].total_bytes*sizeof(float));
	unsigned char *d_band, *d_gs;
	float *d_results;

	// For the rest three images, we need to follow modified Gram-Schmidt routine
	unsigned l, T;
	unsigned long long i;
	for (T = 1; T < 4; T++) 
	{
		phi[T][0] = 0.0f;
		phi[T][1] = 0.0f;
		phi[T][2] = 0.0f;
		phi[T][3] = 0.0f;
		
		for (l = 0; l < T; l++) 
			phi[T][l] = covariance(bands + T, gs + l);

		gpuErrchk(hipMalloc((void**)&d_results, bands[0].total_bytes*sizeof(float)));
		gpuErrchk(hipMalloc((void**)&d_gs, bands[0].total_bytes*sizeof(unsigned char)));


		for (i = 0; i < bands[T].total_bytes; i++) 
			res[i] = (bands[T].data[i] - bands[T].mean);

		gpuErrchk(hipMemcpy( d_results, res, bands[T].total_bytes*sizeof(float), hipMemcpyHostToDevice));

		for (l = 0; l < T; l++)		
		{
			gpuErrchk( hipMemcpy( d_gs, gs[l].data, bands[T].total_bytes*sizeof(unsigned char), hipMemcpyHostToDevice) );
			cuda_GramSchmidt<<< 16, 256>>>(d_results, bands[T].total_bytes, d_gs, phi[T][l]);	

			gpuErrchk( hipPeekAtLastError() );
			gpuErrchk( hipDeviceSynchronize() );

		}


		gpuErrchk( hipMemcpy( res, d_results, bands[T].total_bytes*sizeof(float) , hipMemcpyDeviceToHost ) );
		for (i = 0; i < bands[T].total_bytes; i++) 
			gs[T].data[i] = roundf(res[i]);

		hipFree(d_gs);
		hipFree(d_results);
	}

	free(res);
}

unsigned char get_closest_point(image_t * src, unsigned long long idx, image_t * where) 
{
	int si = idx % src->width;
	int sj = idx / src->width;
	
	int wi = (si / (src->height - 1.0f)) * (where->height - 1.0f);
	int wj = (sj / (src->width - 1.0f))  * (where->width - 1.0f);
	
	return where->data[wj * where->width + wi];
}

// Resize image to new size
void resize(image_t * dst, image_t * src, int new_w, int new_h) 
{
	// Note that we can only resize grayscale images
	if (src->bytes_per_pixel != 1) {
		printf("Resizing is only implemented for grayscale images!\n");
		exit(1);
	}
	
	// First - allocate memory for the dst image
	alloc_image(dst, new_w, new_h, 1);
	
	// Aspect ratio should not be changed
	unsigned long long i;
	
	for (i = 0; i < dst->total_bytes; i++)
		dst->data[i] = get_closest_point(dst, i, src);
}


__global__ void cuda_get_closest_point(unsigned char *dst, int srcHeight, int srcWidth, unsigned long long n, const unsigned char * where, int whereHeight, int whereWidth, float phi) 
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	for(int i = idx; i<n; i += gridDim.x*blockDim.x)
	{
		int si = i % srcWidth;
		int sj = i / srcWidth;
	
		int wi = (si / (srcHeight - 1.0f)) * (whereHeight - 1.0f);
		int wj = (sj / (srcWidth - 1.0f))  * (whereWidth - 1.0f);
	
		dst[i] += phi * where[wj * whereWidth + wi];
	}
}

void GramSchmidtReverseTransformation(image_t * dst, image_t * gs, image_t * bands, float ** phi) 
{
	// Gram-Schmidt imlementation for 4 vectors
	// First gs element is the same as band 0, i.e. artificial low res pan image
	alloc_image(dst + 0, gs[0].width, gs[0].height, gs[0].bytes_per_pixel);
	alloc_image(dst + 1, gs[0].width, gs[0].height, gs[0].bytes_per_pixel);
	alloc_image(dst + 2, gs[0].width, gs[0].height, gs[0].bytes_per_pixel);
	alloc_image(dst + 3, gs[0].width, gs[0].height, gs[0].bytes_per_pixel);
	
	unsigned l, T;
	unsigned char *d_gs, *d_dst;

	float *d_results;

	//float *h_results = (float*)malloc(dst[0].total_bytes*sizeof(float)); 
	//gpuErrchk(cudaMalloc((void**)&d_results, dst[0].total_bytes*sizeof(float)));
	gpuErrchk(hipMalloc((void**)&d_dst, dst[0].total_bytes*sizeof(unsigned char)));
	gpuErrchk(hipMalloc((void**)&d_gs, gs[0].total_bytes*sizeof(unsigned char)));

	for (T = 0; T < 4; T++)
	{

		gpuErrchk(hipMemset( d_dst, (unsigned char)roundf(bands[T].mean), dst[0].total_bytes*sizeof(unsigned char)));

		gpuErrchk(hipMemcpy( d_gs, gs[T].data, gs[T].total_bytes*sizeof(unsigned char), hipMemcpyHostToDevice ));

		cuda_get_closest_point<<< 16, 256>>>(d_dst, dst[0].height, dst[0].width, dst[0].total_bytes, d_gs, gs[T].height,  gs[T].width, 1.0);

		gpuErrchk( hipPeekAtLastError() );
		gpuErrchk( hipDeviceSynchronize() );

		for (l = 0; l < T; l++)
		{
			gpuErrchk( hipMemcpy( d_gs, gs[l].data, gs[T].total_bytes*sizeof(unsigned char), hipMemcpyHostToDevice) );

			cuda_get_closest_point<<< 16, 256>>>(d_dst, dst[0].height, dst[0].width, dst[0].total_bytes, d_gs, gs[l].height,  gs[l].width, phi[T][l]);

			gpuErrchk( hipPeekAtLastError() );
			gpuErrchk( hipDeviceSynchronize() );
		}


		gpuErrchk( hipMemcpy(dst[T].data, d_dst, dst[0].total_bytes*sizeof(unsigned char) , hipMemcpyDeviceToHost ) );

	}

	hipFree(d_gs);	
	hipFree(d_dst);	
	//cudaFree(d_results);
}



void coalesce(image_t * images, image_t * res) 
{
	alloc_image(res, images[1].width, images[1].height, 3);
	
	unsigned long long i;
	for (i = 0; i < images[1].total_bytes; i++) {
		res->data[3 * i + 0] = images[1].data[i];
		res->data[3 * i + 1] = images[2].data[i];
		res->data[3 * i + 2] = images[3].data[i];
	}
}

void stretch(image_t * dst, image_t * src) {
	// No need for dst and src to be of the same size
	eval_stats(dst);
	eval_stats(src);
	
	float gain = src->stddev / dst->stddev;
	float bias = src->mean - gain * dst->mean;
	
	unsigned long long i;
	for (i = 0; i < dst->total_bytes; i++)
		dst->data[i] = roundf(dst->data[i] * gain + bias);
}

int main(int argc, char * argv[]) 
{
    	struct timespec start;
    	struct timespec finish;

	if (argc != 5){
		printf("%s <R-filename1> <G-filename2> <B-filename3> <Pan-filename4>\n", argv[0]);
		return 1;
	}
	
	// Original images
	image_t r, g, b, p;
	
	printf("--- Loading initial images...\n");
	
	read_image(&r, argv[1]); // r
	read_image(&g, argv[2]); // g
	read_image(&b, argv[3]); // b
	read_image(&p, argv[4]); // High res pan band
	
	if ((r.width != g.width) || (r.width != b.width) || (r.height != g.height) || (r.height != b.height)) {
		printf("red, green, blue images are not the same size!\n");
		return 2;
	}
	
	// Create low res pan band
	printf("--- Simulating low res pan band...\n");
	image_t sim_p;
	resize(&sim_p, &p, r.width, r.height);
	
	image_t gs[4];
	image_t bands[4]; // Original bands
	image_t out[4];
	bands[0] = sim_p;
	bands[1] = r;
	bands[2] = g;
	bands[3] = b;
	
	float ** phi;
	phi = (float**)malloc(4 * sizeof(float *));
	unsigned i;
	for (i = 0; i < 4; i++)
		phi[i] = (float*)calloc(4, sizeof(float));
	
	clock_gettime(CLOCK_MONOTONIC, &start);
	printf("--- Executing Gram-Schmidt transformation...\n");
	GramSchmidtTransformation(gs, bands, phi);

	clock_gettime(CLOCK_MONOTONIC, &finish);
	double elapsed;
	elapsed = (finish.tv_sec - start.tv_sec);
	elapsed += (finish.tv_nsec - start.tv_nsec) / 1000000000.0;
	printf("Wall-Clock time passed: %lf \n",elapsed);
	
	printf("--- Stretching high res pan image...\n");
	stretch(&p, gs + 0);
	
	dealloc_image(gs + 0);
	gs[0] = p;
	
	clock_gettime(CLOCK_MONOTONIC, &start);

	printf("--- Executing inverse Gram-Schmidt transformation...\n");
	GramSchmidtReverseTransformation(out, gs, bands, phi);

	clock_gettime(CLOCK_MONOTONIC, &finish);
	elapsed = (finish.tv_sec - start.tv_sec);
	elapsed += (finish.tv_nsec - start.tv_nsec) / 1000000000.0;
	printf("Wall-Clock time passed: %lf \n",elapsed);	


	//write_image(bands+0, "i_in_sim_p.tif");
	//write_image(bands+1, "i_in_r.tif");
	//write_image(bands+2, "i_in_g.tif");
	//write_image(bands+3, "i_in_b.tif");
	
	write_image(gs+0, "i_gs0.tif");
	write_image(gs+1, "i_gs1.tif");
	write_image(gs+2, "i_gs2.tif");
	write_image(gs+3, "i_gs3.tif");
	
	// Deallocate all non-needed images here
	dealloc_image(bands + 0); // sim_p
	dealloc_image(bands + 1); // r
	dealloc_image(bands + 2); // g
	dealloc_image(bands + 3); // b
	dealloc_image(gs + 0); // Modified pan
	dealloc_image(gs + 1); // GS band 1
	dealloc_image(gs + 2); // GS band 2
	dealloc_image(gs + 3); // GS band 3
	
	for (i = 0; i < 4; i++)
		free(phi[i]);
	free(phi);
	
	printf("--- Coalescing image...\n");
	image_t res;
	coalesce(out, &res);
	
	write_image(out+0, "i_out_r.tif");
	write_image(out+1, "i_out_g.tif");
	write_image(out+2, "i_out_b.tif");
	write_image(out+3, "i_out_p.tif");
	
	dealloc_image(out + 0);
	dealloc_image(out + 1);
	dealloc_image(out + 2);
	dealloc_image(out + 3);
	
	printf("--- Writing image...\n");
	write_image(&res, "Gram-Schmidt.tif");
	
	dealloc_image(&res);
	printf("--- Everything is done!\n");
	
	return 0;
}
